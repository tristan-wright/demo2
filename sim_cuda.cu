#include "hip/hip_runtime.h"
#include "sim_cuda.h"



// Error handling function from "cosc3500/cuda/example1-gpu.cu"
void checkError(hipError_t e) {
    if (e != hipSuccess) {
        std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
        abort();
    }
}

__global__
void spins(Surface* lattice) {
    for (int j = 0; j < lattice->size; ++j) {
        for (int k = 0; k < lattice->size; ++k) {
            int coords[2] = {k,j};
            //lattice->calculate_spin(coords);
        }
    }
}


/**
 */
int simulate(Surface lattice) {
    Surface* mLattice;

    for (int i = 0; i < lattice.loops; ++i) {
        lattice.avgEnergy[i] = lattice.calculate_energy();
        lattice.avgMag[i] = lattice.calculate_magnetism();

        checkError(hipMalloc((void **)&mLattice, sizeof(Surface)));
        checkError(hipMemcpy(&mLattice, &lattice, sizeof(Surface), hipMemcpyHostToDevice));

        int Threads = 32;
        int Blocks = (lattice.size+Threads-1)/Threads;

        spins<<<Blocks, Threads>>>(mLattice);

        checkError(hipMemcpy(&lattice, &mLattice, sizeof(Surface), hipMemcpyDeviceToHost));
        hipFree(mLattice);


    }
    return EXIT_SUCCESS;
}

/**
 * Initialisation of clocks to measure the runtime of the different
 * parallelization techniques.
 * @param lattice The Surface object containing the lattice configuration and functions.
 * @return The EXIT_STATUS of the simulation.
 */
int initialise(Surface lattice) {
    lattice.clear();
    lattice.save();
    auto StartTime = std::chrono::high_resolution_clock::now();
    int status = simulate(lattice);
    auto FinishTime = std::chrono::high_resolution_clock::now();
    lattice.complete = true;
    lattice.save();
    auto TotalTime = std::chrono::duration_cast<std::chrono::microseconds>(FinishTime - StartTime);
    cout << lattice.name << ":" << endl;
    cout << "Total time: " << std::setw(12) << TotalTime.count() << " us" << endl;
    return status;
}

/**
 * Main function of the application. Handles boundry and error checking.
 * Will initiate the simulation if everything is correct.
 * @param argc The number of arguments.
 * @param argv The string representation array of the arguments.
 * @return The EXIT_STATUS of the simulation.
 */
int main(int argc, char* argv[]) {
    if (argc < 4 || argc > 5) {
        std::cout << "Usage: ./ising n size temperature {output}" << std::endl;
        return EXIT_FAILURE;
    }

    int n = atoi(argv[1]);
    if (n <= 0) {
        std::cout << "'n' has to be a positive integer" << std::endl;
        return EXIT_FAILURE;
    }

    Surface lattice(argv[0], n, atoi(argv[2]), strtod(argv[3], nullptr));

    if (argc == 5) {
        lattice.out = true;
        lattice.outName = argv[4];
    }

    return initialise(lattice);
}

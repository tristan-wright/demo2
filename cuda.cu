#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

Surface* mLattice;

// Error handling function from "cosc3500/cuda/example1-gpu.cu"
void checkError(hipError_t e) {
    if (e != hipSuccess) {
        std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
        abort();
    }
}

__global__
void spins(Surface* lattice) {
    for (int j = 0; j < lattice.size; ++j) {
        for (int k = 0; k < lattice.size; ++k) {
            int coords[2] = {j,k};
            lattice.calculate_spin(coords);
        }
    }
}


/**
 */
int simulate(Surface lattice) {
    for (int i = 0; i < lattice.loops; ++i) {
        lattice.avgEnergy[i] = lattice.calculate_energy();
        lattice.avgMag[i] = lattice.calculate_magnetism();


        checkError(hipMalloc((void **)&mLattice, sizeof(Surface)));
        checkError(hipMemcpy(mLattice, lattice, sizeof(Surface), hipMemcpyHostToDevice));

        hipFree(mLattice);

        spins(&lattice);
    }
    return EXIT_SUCCESS;
}